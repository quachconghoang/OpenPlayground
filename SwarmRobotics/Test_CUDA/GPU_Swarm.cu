#include "hip/hip_runtime.h"
#pragma warning (disable : 4267)

#include "GPU_Swarm.h"
#include <time.h>
#include <numeric>
#include <limits>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include "thrust/device_ptr.h"
#include "thrust/device_malloc.h"
#include "thrust/device_free.h"

#include "thrust/copy.h"
#include "thrust/fill.h"
#include "thrust/sequence.h"
#include "thrust/sort.h"
#include "thrust/extrema.h"

__global__ void Test_Kernel(float * inGraph, DPSO::Particle * p)
{
	int particleID = threadIdx.x + blockIdx.x*blockDim.x;
	
	DPSO::Particle particle = p[particleID];
	int offsetID = particle.graphOffset;

	p[particleID].psoResult = offsetID;
}

__global__ void CalculateValue_Kernel(float * inGraph, DPSO::Particle * p)
{
	int pID = threadIdx.x + blockIdx.x*blockDim.x;
	DPSO::Particle * _par = &p[pID];
	
	int offsetID = _par->graphOffset;
	int nodeCount = _par->positionSize;
	
	//thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(_par->positionData);
	
	int _psoValue = 0;
	for (int i = 0; i < nodeCount-1; i++)
	{
		int fromNodeID = _par->positionData[i];
		int toNodeID = _par->positionData[i+1];
		float dist = inGraph[offsetID + fromNodeID*nodeCount + toNodeID];
		_psoValue += dist;
	}
	//Calculate last node
	int fromNodeID = _par->positionData[nodeCount-1];
	int toNodeID = _par->positionData[0];
	float dist = inGraph[offsetID + fromNodeID*nodeCount + toNodeID];
	_psoValue += dist;

	_par->psoResult = _psoValue;

	//Update Local-best
	if (_psoValue < _par->bestValue )
	{
		_par->bestValue = _psoValue;
	}
}

namespace DPSO
{
	SwarmCuda::SwarmCuda(int particle_count, float self_trust, float past_trust, float global_trust)
	{
		this->particle_count = particle_count;
		this->self_trust = self_trust;
		this->past_trust = past_trust;
		this->global_trust = global_trust;
		srand(int(time(NULL)));
	}

	void SwarmCuda::read_graph_definition(std::string filename)
	{
		read_graph_files(filename, graphCPU);
		loadGraphToGPU(graphCPU, graphGPU, this->particle_count);
	}

	void SwarmCuda::assign_particle_positions()
	{
		int gSize = graphGPU.num_nodes * graphGPU.num_edgesPerNode;
		int pSize = graphGPU.num_edgesPerNode;
		
		position_Sink.resize(pSize * particle_count);
		best_position_Sink.resize(pSize * particle_count);
		
		best_value = -1;
		best_position.resize(graphGPU.num_nodes);

		for (size_t i = 0; i < particle_count; i++)
		{
			//thrust::fill(position_Sink.begin() + i*pSize, position_Sink.begin() + (i + 1)*pSize, i);
			
			Particle _p;
			_p.pIndex = i;
			_p.graphOffset = i*gSize;
			_p.positionSize = pSize;
			_p.positionOffset = i*pSize;
			_p.psoResult = -1;
			_p.bestValue = std::numeric_limits<float>::max();

			_p.self_trust = this->self_trust;
			_p.past_trust = this->past_trust;
			_p.global_trust = this->global_trust;

			thrust::device_ptr<int> dev_ptr = &position_Sink[i*pSize];
			_p.positionData = thrust::raw_pointer_cast(dev_ptr);
			
			std::vector<int> newPos = shuffle();
			thrust::copy(newPos.begin(), newPos.end(), dev_ptr);

			gpuParticles.push_back(_p);
		}

		/*for (int i = 0; i < particle_count; i++)
		{
			Particle _p = gpuParticles[i];
			thrust::device_ptr<int> dev_ptr(_p.positionData);
			thrust::device_vector<int> dev_vec(dev_ptr, dev_ptr + pSize);

			std::cout << "Particle : " << i << std::endl;
			for (int j = 0; j < pSize; j++)
			{
				std::cout << dev_vec[j] << " ";
			}
			std::cout << std::endl;
		}*/

		std::cout << "Assigned! \n";

		int threadsPerBlock = particle_count;
		float * raw_graph_ptr = thrust::raw_pointer_cast(graphGPU.graphData.data());
		DPSO::Particle * parts = thrust::raw_pointer_cast(gpuParticles.data());
		CalculateValue_Kernel <<< 1, threadsPerBlock >> >(raw_graph_ptr, parts);
		std::cout << "First iteration \n";

		/// - TRACING
		for (int j = 0; j < particle_count; j++)
		{
			Particle _p = gpuParticles[j];
			std::cout << "P0[" << j << "] = " << _p.psoResult << std::endl;
		}

		/// - Can be replace with shared memory in Kernel ???
		int64_t minEle = thrust::min_element(gpuParticles.begin(), gpuParticles.end()) - gpuParticles.begin();
		Particle tPar = gpuParticles[minEle];

		thrust::copy(position_Sink.begin() + tPar.positionOffset,
			position_Sink.begin() + tPar.positionOffset + tPar.positionSize,
			best_position.begin());

		std::cout << "Min element P[" << minEle << "] = " << tPar.bestValue << std::endl;
	}

	std::vector<int> SwarmCuda::shuffle()
	{
		/// shuffle 0->NodeSize
		//Knuth-Fisher-Yates shuffle
		int pSize = graphGPU.num_nodes;
		std::vector<int> newPos(pSize);
		std::iota(newPos.begin(), newPos.end(), 0);

		for (int i = (int)newPos.size() - 1; i > 0; i--)
		{
			int n = rand() % (i + 1);
			int tmp = newPos[i];
			newPos[i] = newPos[n];
			newPos[n] = tmp;
		}
		return newPos;
	}

	void SwarmCuda::testSwarmAction()
	{
		std::cout << "Testing ! \n" << std::endl;
		int threadsPerBlock = this->particle_count;
		int blocksPerGrid = 1;

		float * raw_graph_ptr = thrust::raw_pointer_cast(graphGPU.graphData.data());

		DPSO::Particle * parts = thrust::raw_pointer_cast(gpuParticles.data());
		Test_Kernel <<< blocksPerGrid, threadsPerBlock >>>(raw_graph_ptr, parts);

		for (int i = 0; i < gpuParticles.size();i++)
		{
			DPSO::Particle p = gpuParticles[i];
			std::cout << "R[" << i << "] = " << p.psoResult << std::endl;
		}

		//size_t max_ele = thrust::max_element(gpuParticles.begin(), gpuParticles.end()) - gpuParticles.begin();
		//std::cout << "Max element = " << max_ele << std::endl;
	}
}

void read_graph_files(std::string filename, std::vector<GraphNode> & graphData)
{
	std::ifstream graph_file;
	graph_file.open(filename.c_str());
	if (graph_file.is_open()){
		int totalNode;
		graph_file >> totalNode;
		std::cout << "Reading nodes :\n";

		for (int i = 0; i < totalNode; i++)
		{
			GraphNode n;
			n.cost_to.resize(totalNode);
			graph_file >> n.index;
			graph_file >> n.x;	graph_file >> n.y;	graph_file >> n.z;

			for (int j = 0; j < totalNode; j++){
				graph_file >> n.cost_to[j];
			}
			graphData.push_back(n);
		}
		graph_file.close();
		std::cout << totalNode << " nodes were read ... \n";
	}
	else{
		std::cout << "Could not open file" << std::endl;	throw(-1);
	}
}

void loadGraphToGPU(std::vector<GraphNode> & graphCPU, GraphGPU & graphGPU, int numParticles)
{
	graphGPU.num_nodes = (int)graphCPU.size();
	graphGPU.num_edgesPerNode = (int)graphCPU.size();
	graphGPU.num_particles = numParticles;

	int edgeSize = graphGPU.num_edgesPerNode;
	int partSize = graphGPU.num_nodes * graphGPU.num_edgesPerNode;

	std::vector<float> fullData(partSize);
	for (int i = 0; i < graphCPU.size(); i++)
	{
		std::copy(graphCPU[i].cost_to.begin(),
			graphCPU[i].cost_to.end(),
			fullData.begin() + i*edgeSize);
	}

	/// - Load graph to Thrust vector
	graphGPU.graphData.clear();
	graphGPU.graphData.resize(partSize*numParticles);
	for (int i = 0; i < numParticles; i++)
	{
		thrust::copy(fullData.begin(), fullData.end(),
			graphGPU.graphData.begin() + i*partSize);
	}

	/*for (int n = 0; n < numParticles; n++)
	{
		int part_offset = n*partSize;
		std::cout << "NODE = " << n << ": \n";
		for (int i = 0; i < graphGPU.num_nodes; i++)
		{
			for (int j = 0; j < graphGPU.num_edgesPerNode; j++)
			{
				int idX = part_offset + i*graphGPU.num_edgesPerNode + j;
				std::cout << graphGPU.graphData[idX] << " ";
			}
			std::cout << std::endl;
		}
	}*/
}
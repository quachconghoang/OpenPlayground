#include "hip/hip_runtime.h"
#pragma warning (disable : 4267)

#include "GPU_Swarm.h"
#include <time.h>
#include <numeric>
#include <limits>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include "thrust/device_ptr.h"
#include "thrust/device_malloc.h"
#include "thrust/device_free.h"

#include "thrust/copy.h"
#include "thrust/fill.h"
#include "thrust/sequence.h"
#include "thrust/sort.h"
#include "thrust/extrema.h"

#include "opencv2/opencv.hpp"

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

__device__ void calculateValueParticle(DPSO::Particle * _par)
{
	int nodeCount = _par->positionSize;

	float _psoValue = 0;
	for (int i = 0; i < nodeCount - 1; i++)
	{
		int fromNodeID = _par->positionData[i];
		int toNodeID = _par->positionData[i + 1];
		float dist = _par->graphData[fromNodeID*nodeCount + toNodeID];
		_psoValue += dist;
	}
	//Calculate last node
	int fromNodeID = _par->positionData[nodeCount - 1];
	int toNodeID = _par->positionData[0];
	float dist = _par->graphData[fromNodeID*nodeCount + toNodeID];
	_psoValue += dist;

	_par->psoResult = _psoValue;

	//Update Local-best
	if (_psoValue < _par->bestValue)
	{
		_par->bestValue = _psoValue;
		for (int i = 0; i < _par->positionSize; i++)
		{
			_par->bestPosition[i] = _par->positionData[i];
		}
	}
}

__device__ float edge_length(int locID, int nodeCount, float * _graph)
{
	int next_locID = (locID == (nodeCount - 1)) ? 0 : locID + 1;
	return _graph[locID*nodeCount + next_locID];
}

__device__ float edge_length_swap(int i, int j, int nodeCount, float * _graph)
{
	int next_i = (i == nodeCount - 1) ? 0 : i + 1;  // If index = (node_number - 1)  -> last node -> next node is set to be the start node
	int next_j = (j == nodeCount - 1) ? 0 : j + 1;

	float edge_length_i = _graph[i*nodeCount + next_i];
	float edge_length_j = _graph[j*nodeCount + next_j];

	return (edge_length_i + edge_length_j);
}

__global__ void CalculateValue_Kernel(DPSO::Particle * p)
{
	int pID = threadIdx.x + blockIdx.x*blockDim.x;
	DPSO::Particle * _par = &p[pID];
	calculateValueParticle(_par);
}

__global__ void Moving_Kernel(DPSO::Particle * p, int bestParticleID)
{
	/// 0. Load best position to array
	__shared__ int best[1024];
	DPSO::Particle * _bestPar = &p[bestParticleID];
	for (int i = 0; i < _bestPar->positionSize;i++)
	{
		best[i] = _bestPar->bestPosition[i];
	}
	//int pcID = threadIdx.x + blockIdx.x*blockDim.x;
	//DPSO::Particle * _bestPar = &p[bestParticleID];
	//int positionSize = _bestPar->positionSize;
	//int step = 0;
	//int cpThreadLoc;
	//while ((cpThreadLoc = pcID + step*NUM_PARTICLES) < positionSize)
	//{
	//	best[cpThreadLoc] = _bestPar->bestPosition[cpThreadLoc];
	//	step++;
	//}
	__syncthreads();
	
	/// 1. Calculate new velocities from best-position & trust values (c2 & c3)
	int pID = threadIdx.x + blockIdx.x*blockDim.x;
	DPSO::Particle * _par = &p[pID];

	//calculate_Velocities(_par, &best[0]);

	int posSize = _par->positionSize;
	int _velocitySize = 0;

	// 1a. Get velocity from C2: past-trust
	int c2Size = ceilf(posSize *_par->past_trust);
	for (int i = 0; i < c2Size; i++)
	{
		int look_for_ID = _par->positionData[i];
		int found_at = -1;
		for (int j = 0; j < posSize; j++)
		{
			if (look_for_ID == _par->positionData[j])	{	found_at = j;	break;	}
		}

		if ((i != found_at) && (found_at != -1))
		{
			_par->velocity[_velocitySize].from = i;
			_par->velocity[_velocitySize].to = found_at;
			_velocitySize++;
		}
	}

	// 1b. Get velocity from C3: global-trust
	int v_offset = _velocitySize;
	int c3Size = ceilf(posSize*_par->global_trust);
	for (int i = 0; i < c3Size; i++)
	{
		int look_for_ID = _par->positionData[i];
		int found_at = -1;
		for (int j = 0; j < posSize; j++)
		{
			if (look_for_ID == best[j])	{ found_at = j;	break; }
		}

		if ((i != found_at) && (found_at != -1))
		{
			int vID = v_offset + i;
			_par->velocity[vID].from = i;
			_par->velocity[vID].to = found_at;
			_velocitySize++;
		}
	}

	_par->velocitySize = _velocitySize;
	//printf("Thread: %d - c1 velocity_size %d \n", pID, _velocitySize);

	/// 2. Calculate best position from new velocities
	//		2a. Check all velocity
	//		2b. Swap node positions
	for (int i = 0; i < _velocitySize; i++)
	{
		DPSO::D_Vec2i swapVal = _par->velocity[i];
		int tmp = _par->positionData[swapVal.from];
		_par->positionData[swapVal.from] = _par->positionData[swapVal.to];
		_par->positionData[swapVal.to] = tmp;
	}
	_par->velocitySize = 0;

	/// 3. Two opt
	float best_cost_change = 0;
	int i_best = -1;
	int j_best = -1;
	int node_count = _par->positionSize;

	for (int i = 0; i < node_count - 2; i++){
		float edge_length_i = edge_length(i,node_count,_par->graphData);

		for (int j = i + 2; j < node_count; j++){
			float old_cost = edge_length_i + edge_length(j, node_count, _par->graphData);;
			float new_cost = edge_length_swap(i, j, node_count, _par->graphData);
			float cost_change = new_cost - old_cost;

			if (cost_change < best_cost_change)	{
				best_cost_change = cost_change;
				i_best = i;
				j_best = j;
			}
		}
	}
	/// Swap edges;
	int middle = (j_best - i_best) / 2;
	for (int k = 0; k < middle; k++) {
		int _from = i_best + k + 1;
		int _to = j_best + k;
		
		/// Swapping
		int tmp = _par->positionData[_from];
		_par->positionData[_from] = _par->positionData[_to];
		_par->positionData[_to] = tmp;
	}

	/// 4. Calculate Values
	calculateValueParticle(_par);
}

namespace DPSO
{
	SwarmCuda::SwarmCuda(int particle_count, float self_trust, float past_trust, float global_trust)
	{
		this->particle_count = particle_count;
		this->self_trust = self_trust;
		this->past_trust = past_trust;
		this->global_trust = global_trust;
		srand(unsigned int(time(NULL)));
	}

	void SwarmCuda::read_graph_definition(std::string filename)
	{
		read_graph_files(filename, graphCPU);
		loadGraphToGPU(graphCPU, graphGPU, this->particle_count);
	}

	void SwarmCuda::assign_particle_positions()
	{
		int gSize = graphGPU.num_nodes * graphGPU.num_edgesPerNode;
		int pSize = graphGPU.num_nodes;
		
		position_Sink.resize(pSize * particle_count);
		best_position_Sink.resize(pSize * particle_count);
		velocity_Sink.resize(pSize*particle_count);

		best_value = -1;

		for (size_t i = 0; i < particle_count; i++)
		{	
			//thrust::fill(position_Sink.begin() + i*pSize, position_Sink.begin() + (i + 1)*pSize, -1);
			//thrust::fill(best_position_Sink.begin() + i*pSize, best_position_Sink.begin() + (i + 1)*pSize, -1);
			//DPSO::D_Vec2i v_i; v_i.from = 0; v_i.to = 0;
			//thrust::fill(velocity_Sink.begin() + i*pSize, velocity_Sink.begin() + (i + 1)*pSize, v_i);

			Particle _p;
			_p.pIndex = i;
			_p.graphOffset = i*gSize;
			_p.positionSize = pSize;
			_p.positionOffset = i*pSize;
			_p.psoResult = -1;
			_p.bestValue = std::numeric_limits<float>::max();
			_p.velocitySize = 0;

			_p.self_trust = this->self_trust;
			_p.past_trust = this->past_trust;
			_p.global_trust = this->global_trust;

			thrust::device_ptr<int> dev_ptr = &position_Sink[i*pSize];
			_p.positionData = thrust::raw_pointer_cast(dev_ptr);
			_p.bestPosition = thrust::raw_pointer_cast(&best_position_Sink[i*pSize]);
			_p.velocity = thrust::raw_pointer_cast(&velocity_Sink[i*pSize]);
			_p.graphData = thrust::raw_pointer_cast(&graphGPU.graphData[i*gSize]);
			
			std::vector<int> newPos = shuffle();
			thrust::copy(newPos.begin(), newPos.end(), dev_ptr);

			gpuParticles.push_back(_p);
		}

		//showParticleData();
		std::cout << "Assigned! \n";

		int threadsPerBlock = particle_count;
		DPSO::Particle * parts = thrust::raw_pointer_cast(gpuParticles.data());
		CalculateValue_Kernel <<< 1,threadsPerBlock >>>(parts);

		std::cout << "First iteration \n";
		bestParticleNum = int(thrust::min_element(gpuParticles.begin(), gpuParticles.end()) - gpuParticles.begin());
		Particle tPar = gpuParticles[bestParticleNum];
		best_value = tPar.bestValue;
		std::cout << " - P[" << bestParticleNum << "] = " << best_value << std::endl;

		//move_particle();
		//showParticleData();
	}

	bool SwarmCuda::move_particle()
	{
		//float previous_best = this->best_value;
		bool best_changed = false;

		int threadsPerBlock = particle_count;
		//float * raw_graph_ptr = thrust::raw_pointer_cast(graphGPU.graphData.data());
		DPSO::Particle * parts = thrust::raw_pointer_cast(gpuParticles.data());
		Moving_Kernel << < 1, threadsPerBlock >> >(parts, bestParticleNum);
		//hipDeviceSynchronize();
		int new_BestParticleNum = int(thrust::min_element(gpuParticles.begin(), gpuParticles.end()) - gpuParticles.begin());
		Particle tPar = gpuParticles[new_BestParticleNum];
		//std::cout << " - P[" << bestParticleNum << "] = " << best_value << std::endl;
		if (this->best_value > tPar.bestValue)
		{
			this->best_value = tPar.bestValue;
			this->bestParticleNum = new_BestParticleNum;

			best_changed = true;
		}
		return best_changed;
	}

	void SwarmCuda::solve()
	{
		int moves_since_best_changed = 0;
		int64 startTime = cv::getCPUTickCount();
		int iterations = 0;
		int stoppingCount = 20;
		while (moves_since_best_changed <= stoppingCount){
			bool best_changed = false;
			iterations++;
			if (moves_since_best_changed < stoppingCount){  // if cost changing steadily
				best_changed = move_particle();
			}

			if (!best_changed){
				moves_since_best_changed++;
			}
			else{
				moves_since_best_changed = 0;
			}
			std::cout << "Best value so far: " << best_value << std::endl;
		}

		int64 stopTime = cv::getCPUTickCount();
		double timeR = (stopTime - startTime) / cv::getTickFrequency();
		std::cout << "GPU Runtime = " << timeR << "\n Average = " << timeR/iterations;
	}

	std::vector<int> SwarmCuda::shuffle()
	{
		/// shuffle 0->NodeSize
		//Knuth-Fisher-Yates shuffle
		int pSize = graphGPU.num_nodes;
		std::vector<int> newPos(pSize);
		std::iota(newPos.begin(), newPos.end(), 0);

		for (int i = (int)newPos.size() - 1; i > 0; i--)
		{
			int n = rand() % (i + 1);
			int tmp = newPos[i];
			newPos[i] = newPos[n];
			newPos[n] = tmp;
		}
		return newPos;
	}

	void SwarmCuda::showParticleData()
	{
		int pSize = graphGPU.num_edgesPerNode;
		//int gSize = graphGPU.num_edgesPerNode*graphGPU.num_nodes;

		for (int i = 0; i < particle_count; i++)
		{
			Particle _p = gpuParticles[i];
			thrust::device_ptr<int> dev_ptr(_p.positionData);
			thrust::device_vector<int> dev_vec(dev_ptr, dev_ptr + pSize);

			std::cout << "Particle : " << i << std::endl;
			std::cout << " - Value : " << _p.psoResult << std::endl;
			std::cout << " - Best : " << _p.bestValue << std::endl;
			std::cout << " - Nodes: ";
			for (int j = 0; j < pSize; j++){	std::cout << dev_vec[j] << " ";	}
			std::cout << std::endl;

			/*thrust::device_ptr<float> g_ptr(_p.graphData);
			thrust::device_vector<float> g_vec(g_ptr, g_ptr + gSize);
			std::cout << " - Graph: \n";
			for (int j = 0; j < pSize; j++)
			{
				for (int k = 0; k < pSize; k++){std::cout << g_vec[j*pSize+k] << " ";}
				std::cout << std::endl;
			}
			std::cout << std::endl;*/

			thrust::device_ptr<int> ptr2(_p.bestPosition);
			thrust::device_vector<int> vec2(ptr2, ptr2 + pSize);
			std::cout << " - BestPosition: ";
			for (int j = 0; j < pSize; j++){	std::cout << vec2[j] << " ";	}
			std::cout << std::endl;

			/*thrust::device_ptr<DPSO::D_Vec2i> ptr3(_p.velocity);
			thrust::device_vector<DPSO::D_Vec2i> vec3(ptr3, ptr3 + pSize);
			std::cout << " - VelocitySize : " << _p.velocitySize << std::endl;
			std::cout << " - Velocity: ";
			for (int j = 0; j < pSize; j++)
			{
				DPSO::D_Vec2i v = vec3[j];
				std::cout << v.from << "-" << v.to << " ";
			}
			std::cout << std::endl;*/
		}
	}
}

void read_graph_files(std::string filename, std::vector<GraphNode> & graphData)
{
	std::ifstream graph_file;
	graph_file.open(filename.c_str());
	if (graph_file.is_open()){
		int totalNode;
		graph_file >> totalNode;
		std::cout << "Reading nodes :\n";

		for (int i = 0; i < totalNode; i++)
		{
			GraphNode n;
			n.cost_to.resize(totalNode);
			graph_file >> n.index;
			graph_file >> n.x;	graph_file >> n.y;	graph_file >> n.z;

			for (int j = 0; j < totalNode; j++){
				graph_file >> n.cost_to[j];
			}
			graphData.push_back(n);
		}
		graph_file.close();
		std::cout << totalNode << " nodes were read ... \n";
	}
	else{
		std::cout << "Could not open file" << std::endl;	throw(-1);
	}
}

void loadGraphToGPU(std::vector<GraphNode> & graphCPU, GraphGPU & graphGPU, int numParticles)
{
	graphGPU.num_nodes = (int)graphCPU.size();
	graphGPU.num_edgesPerNode = (int)graphCPU.size();
	graphGPU.num_particles = numParticles;

	int edgeSize = graphGPU.num_edgesPerNode;
	int partSize = graphGPU.num_nodes * graphGPU.num_edgesPerNode;

	std::vector<float> fullData(partSize);
	for (int i = 0; i < graphCPU.size(); i++)
	{
		std::copy(graphCPU[i].cost_to.begin(),
			graphCPU[i].cost_to.end(),
			fullData.begin() + i*edgeSize);
	}

	/// - Load graph to Thrust vector
	graphGPU.graphData.clear();
	graphGPU.graphData.resize(partSize*numParticles);
	for (int i = 0; i < numParticles; i++)
	{
		thrust::copy(fullData.begin(), fullData.end(),
			graphGPU.graphData.begin() + i*partSize);
	}

	/*for (int n = 0; n < numParticles; n++)
	{
		int part_offset = n*partSize;
		std::cout << "NODE = " << n << ": \n";
		for (int i = 0; i < graphGPU.num_nodes; i++)
		{
			for (int j = 0; j < graphGPU.num_edgesPerNode; j++)
			{
				int idX = part_offset + i*graphGPU.num_edgesPerNode + j;
				std::cout << graphGPU.graphData[idX] << " ";
			}
			std::cout << std::endl;
		}
	}*/
}
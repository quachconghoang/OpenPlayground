#include "hip/hip_runtime.h"
﻿#include "GraphGPU.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include "thrust/device_ptr.h"
#include "thrust/device_malloc.h"
#include "thrust/device_free.h"

#include "thrust/copy.h"
#include "thrust/fill.h"
#include "thrust/sequence.h"
#include "thrust/sort.h"

void test_Thrust()
{
	thrust::device_vector<int> D(10, 0);
	thrust::sequence(D.begin(), D.end());
	for (int i = 0; i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
}

void test_Thrust_v1()
{
	// H has storage for 4 integers
	thrust::host_vector<int> H(4);
	// initialize individual elements
	H[0] = 14;
	H[1] = 20;
	H[2] = 38;
	H[3] = 46;
	std::cout << "H has size " << H.size() << std::endl;
	for (int i = 0; i < H.size(); i++)
		std::cout << "H[" << i << "] = " << H[i] << std::endl;

	H.resize(2);
	std::cout << "H now has size " << H.size() << std::endl;
	thrust::device_vector<int> D = H;
	D[0] = 99;
	D[1] = 88;

	for (int i = 0; i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
}

void test_Thrust_v2()
{
	// initialize all ten integers of a device_vector to 1
	thrust::device_vector<int> D(10, 1);
	// set the first seven elements of a vector to 9
	thrust::fill(D.begin(), D.begin() + 7, 9);
	// initialize a host_vector with the first five elements of D
	thrust::host_vector<int> H(D.begin(), D.begin() + 5);
	// set the elements of H to 0, 1, 2, 3, ...
	thrust::sequence(H.begin(), H.end());
	// copy all of H back to the beginning of D
	thrust::copy(H.begin(), H.end(), D.begin());
	// print D
	for (int i = 0; i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
}

void test_Thrust_v3()
{
	// generate 16M random numbers on the host
	thrust::host_vector<int> h_vec(1 << 8);
	thrust::generate(h_vec.begin(), h_vec.end(), rand);
	// transfer data to the device
	thrust::device_vector<int> d_vec = h_vec;
	// sort data on the device
	thrust::sort(d_vec.begin(), d_vec.end());
	// transfer data back to host
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	for (int i = 0; i < d_vec.size(); i++)
		std::cout << "D[" << i << "] = " << d_vec[i] << std::endl;
}


__global__ void my_kernel(int * a)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	a[i] = i;
}

void test_Thrust_toCUDA()
{
	size_t N = 256;
	int threadsPerBlock = 128;
	int blocksPerGrid = 2;

	/*thrust::device_ptr<int> dev_ptr = thrust::device_malloc<int>(N);
	int * raw_ptr_x = thrust::raw_pointer_cast(dev_ptr);
	hipMemset(raw_ptr_x, 0, N * sizeof(int));
	thrust::device_free(dev_ptr);*/

	thrust::device_vector<int> d_vec(N);

	// note: d_vec.data() returns a device_ptr
	int * raw_ptr = thrust::raw_pointer_cast(d_vec.data());

	//hipMemset(raw_ptr, 10, N * sizeof(int));
	
	my_kernel << < blocksPerGrid, threadsPerBlock >> >(raw_ptr);

	for (int i = 0; i < d_vec.size(); i++)
		std::cout << "D[" << i << "] = " << d_vec[i] << std::endl;
}

void test_Thrust_fromCUDA()
{
	size_t N = 10;

	// obtain raw pointer to device memory
	int * raw_ptr;
	hipMalloc((void **)&raw_ptr, N * sizeof(int));

	// wrap raw pointer with a device_ptr 
	thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(raw_ptr);

	// use device_ptr in Thrust algorithms
	thrust::fill(dev_ptr, dev_ptr + N, (int)0);

	// access device memory transparently through device_ptr
	dev_ptr[0] = 1;

	// free memory
	hipFree(raw_ptr);
}

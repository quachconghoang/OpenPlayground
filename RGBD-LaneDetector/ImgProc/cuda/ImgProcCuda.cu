#include "hip/hip_runtime.h"
#include "ImgProcCuda.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "device_types.h"
#include "hip/device_functions.h"
#include ""

#include "math_functions.h"
#include "hip/hip_math_constants.h"

#define __cds_device__ __device__ __forceinline__

int divUp(int a, int b){ return (a + b - 1) / b; }

__cds_device__ float dot(const float3& v1, const float3& v2)
{
	return __fmaf_rn(v1.x, v2.x, __fmaf_rn(v1.y, v2.y, v1.z*v2.z));
}

__cds_device__ float3 cross(const float3& v1, const float3& v2)
{
	return make_float3(v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x);
}

__cds_device__ float3 operator+(const float3& v1, const float3& v2)
{
	return make_float3(v1.x + v2.x, v1.y + v2.y, v1.z + v2.z);
}

__cds_device__ float3 operator-(const float3& v1, const float3& v2)
{
	return make_float3(v1.x - v2.x, v1.y - v2.y, v1.z - v2.z);
}

// multiply
__cds_device__ float3 operator*(const float3& v1, const float3& v2)
{
	return make_float3(v1.x * v2.x, v1.y * v2.y, v1.z * v2.z);
}

__cds_device__ float3 operator*(float3 a, float s)
{
	return make_float3(a.x * s, a.y * s, a.z * s);
}

__cds_device__ float3 operator*(float s, float3 a)
{
	return make_float3(a.x * s, a.y * s, a.z * s);
}

// normalize
__cds_device__ float3 normalize(const float3& v)
{
	return  rsqrt(dot(v, v)) * v;
}



__global__ void kernel_convert_Depth_To_Point3f(cv::cuda::PtrStep<ushort> _depth, const ImgProc3D::Intr cam, cv::cuda::PtrStep<float3> _point3f)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float d = float(_depth(y,x)) / cam.scale;
	if (d == 0)
		_point3f(y, x) = { HIP_NAN_F, HIP_NAN_F, HIP_NAN_F };
	else
		_point3f(y, x) = { (x - cam.cx) * d / cam.fx, (y - cam.cy) * d / cam.fy, d };
	return;
}

__global__ void kernel_convert_XYZ_To_Normals(int rows, int cols, cv::cuda::PtrStep<float3> vmap, cv::cuda::PtrStep<float3> nmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= cols || y >= rows)
		return;

	if (x == cols - 1 || y == rows - 1)
	{
		nmap(y, x) = { HIP_NAN_F, HIP_NAN_F, HIP_NAN_F }; /*HIP_NAN_F*/
		return;
	}
	float3 v00, v01, v10;

	v00 = vmap(y ,x);
	v01 = vmap(y, x+1);
	v10 = vmap(y+1, x);

	if (!isnan(v00.x) && !isnan(v01.x) && !isnan(v10.x))
	{
		nmap(y, x) = normalize(cross(v01 - v00, v10 - v00));
	}
	else
	{
		//nmap(y, x) = { HIP_NAN_F, HIP_NAN_F, HIP_NAN_F };
		nmap(y, x).x = HIP_NAN_F;
		nmap(y, x).y = HIP_NAN_F;
		nmap(y, x).z = HIP_NAN_F;
	}
}

__global__ void kernel_GenGridMap2D(cv::cuda::PtrStep<float3> _point3f, cv::cuda::PtrStep<uchar3> _rgb,
	float4 pModel,
	float3 pOrg, float3 e_1, float3 e_2,
	cv::cuda::PtrStep<uchar3> _map,
	cv::cuda::PtrStep<uchar> _objMask)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float3 p = _point3f(y, x);

	float3 p_new = p - pOrg;
	float p_x_new = dot(e_1, p_new);
	float p_y_new = dot(e_2, p_new);

	int new_x = int(LANE_MAP_SIZE / 2 + p_x_new * LANE_MAP_SCALE);
	int new_y = int(LANE_MAP_SIZE - p_y_new * LANE_MAP_SCALE);

	if (fabs(pModel.x*p.x + pModel.y*p.y + pModel.z*p.z + pModel.w) < 0.1f)
	{
		if (new_x > 0 && new_x < LANE_MAP_SIZE  &&  new_y > 0 && new_y < LANE_MAP_SIZE)
		{
			_map(new_y, new_x) = _rgb(y, x);
		}
	}
	else
	{
		if (!isnan(p.z) && p.z < OBJ_DISTANCE_THRESHOLD){
			_objMask(y, x) = 255;
		}
		
		//if (new_x > 0 && new_x < LANE_MAP_SIZE  &&  new_y > 0 && new_y < LANE_MAP_SIZE)
		//{
		//	_map(new_y, new_x).x = 0;
		//	_map(new_y, new_x).y = 0;
		//	_map(new_y, new_x).z = 255;
		//}
	}

	return;
}

void ImgProc3D::convertTo_Point3fMap(cv::cuda::GpuMat & depth, const ImgProc3D::Intr camInfo, cv::cuda::GpuMat & xyzMat)
{
	cv::Size sz = depth.size();
	dim3 block(32, 16);
	dim3 grid(divUp(sz.width, block.x), divUp(sz.height, block.y));
	kernel_convert_Depth_To_Point3f << <grid, block >> >(depth, camInfo, xyzMat);
}

void ImgProc3D::convertTo_NormalsMap(cv::cuda::GpuMat & xyzMat, cv::cuda::GpuMat & normalMap)
{
	cv::Size sz = xyzMat.size();
	dim3 block(32, 16);
	dim3 grid(divUp(sz.width, block.x), divUp(sz.height, block.y));
	kernel_convert_XYZ_To_Normals << <grid, block >> >(sz.height, sz.width, xyzMat, normalMap);
}

void ImgProc3D::genPlane2DMap(cv::cuda::GpuMat & xyzMat, cv::cuda::GpuMat & rgbMat, cv::Vec4f planeModel, cv::cuda::GpuMat & laneMap, cv::cuda::GpuMat & objMask)
{
	cv::Point3f planeNormal(planeModel[0], planeModel[1], planeModel[2]);
	cv::Point3f e_1 = planeNormal.cross(cv::Point3f(0, 0, 1));
	cv::Point3f e_2 = -planeNormal.cross(e_1);
	cv::Point3f planeOrg = -planeModel[3] * planeNormal;

	cv::Size sz = xyzMat.size();
	dim3 block(32, 16);
	dim3 grid(divUp(sz.width, block.x), divUp(sz.height, block.y));

	float4 _model = { planeModel[0], planeModel[1], planeModel[2], planeModel[3] };
	float3 _pOrg = { planeOrg.x, planeOrg.y, planeOrg.z };
	float3 _e_1 = { e_1.x, e_1.y, e_1.z };
	float3 _e_2 = { e_2.x, e_2.y, e_2.z };
	kernel_GenGridMap2D << <grid, block >> >(xyzMat, rgbMat, _model, _pOrg, _e_1, _e_2, laneMap, objMask);
}